
#include <hip/hip_runtime.h>
__global__
void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

//function to compare and check results
void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);
  
  // What happens when unified memory is accessed only by the GPU?
  int threadsPerBlock = 32 << 2; //max: 1024
  int blocksPerGrid = 32;
  
  int choice = 4;
  if(choice == 1)
  {
   //print("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
   deviceKernel<<<blocksPerGrid, threadsPerBlock>>>(a, N);
  }
  if(choice == 2){
   hostFunction(a, N);
  } 
  if(choice == 3){
   deviceKernel<<<blocksPerGrid, threadsPerBlock>>>(a, N);
   hostFunction(a, N); 
  }
  if(choice == 4){
   hostFunction(a, N);
   deviceKernel<<<blocksPerGrid, threadsPerBlock>>>(a, N);
  }

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU?
   * What happens when unified memory is accessed only by the CPU?
   * What happens when unified memory is accessed first by the GPU then the CPU?
   * What happens when unified memory is accessed first by the CPU then the GPU?
   *
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiement, and then verify by running `nvprof`.
   */

  hipFree(a);
}

