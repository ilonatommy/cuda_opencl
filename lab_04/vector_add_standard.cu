
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  //printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
  printf("Running the standard vector add using managed memory with %d streaming multiprocessors.",numberOfSMs );
  const long long int N = 2<<31;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;
  printf("Grid layout: %d threads per block, %d blocks per grid.", threadsPerBlock, numberOfBlocks);

  hipError_t addVectorsErr;
  hipError_t asyncErr;
  for(int i=0; i < 5; i++)
  {
     initWith(3, a, N);
     initWith(4, b, N);
     initWith(0, c, N);

     addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
  
     addVectorsErr = hipGetLastError();
     if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

     asyncErr = hipDeviceSynchronize();
     if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

     checkElementsAre(7, c, N);
  }
  hipFree(a);
  hipFree(b);
  hipFree(c);
}

