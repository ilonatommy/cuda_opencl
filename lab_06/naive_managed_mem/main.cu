
/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixMultiplicationKernel(float* M, float* N, float* P, int Width) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if ((Row < Width) && (Col < Width)) {
		float Pvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < Width; ++k) {
			Pvalue += M[Row*Width+k]*N[k*Width+Col];
		}
		P[Row*Width+Col] = Pvalue;
	}
}

void matrixMultiplication(float *M, float *N, float *P, int Width){

    // declare the number of blocks per grid and the number of threads per block
    int th = Width;
    int bl= 1;
    dim3 threadsPerBlock(th,th);
    dim3 blocksPerGrid(bl,bl);
    printf("Kernel started: %d blocks, %d threads.\n", bl,th);
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, P, Width);
}

void PrintMatrix(float* M, int Width)
{
	for(int i = 0; i < Width; i++)
	{
		for(int j = 0; j < Width; j++)
			printf("%f  ",M[i*Width+j]);
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = 100;
    	int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================Shared Memory============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float *M;
	err = hipMallocManaged((void**)&M,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate M matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float *N;
	err = hipMallocManaged((void**)&N,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate N matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float *P;
	err = hipMallocManaged((void**)&P,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate P matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

	//initialisation:
	for(int i=0; i<num_of_elements; i++)
	{
	  M[i] = rand()/(float)RAND_MAX;
	  N[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialisation finished.\n");
    
    //calculations:
    matrixMultiplication(M, N, P, matrix_size);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    
    //==========================TEST===============================================
	PrintMatrix(M, matrix_size);
	PrintMatrix(N, matrix_size);
	PrintMatrix(P, matrix_size);	

	for(int i = 0; i < matrix_size; i++)
	{
		for(int j = 0; j < matrix_size; j++)
		{
			float tmp = 0;
			for(int k = 0; k < matrix_size; k++)
				tmp += M[i*matrix_size + k] * N[k*matrix_size + j];
			//debug line:
			//printf("%f ",tmp);
			if(fabs(tmp - P[i*matrix_size + j]) > 1e-3)
			{
				fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
					i,j,tmp,P[i*matrix_size + j]);
				exit(EXIT_FAILURE);
			}
		}
    	}

	printf("Test PASSED\n");
    
    // Free device global memory
    err = hipFree(M);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(N);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix N (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(P);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix P (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Done\n");
    return 0;    
	
}

