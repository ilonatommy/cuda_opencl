
/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *  Author: cuda-s01
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const int TILE_WIDTH = 2;
const int MATRIX_SIZE =8;


__global__ void matrixMultiplicationKernel(float* M, float* N, float* P, int Width) {
        // Calculate the row index of the P element and M
        int Row = blockIdx.y*blockDim.y+threadIdx.y;
        // Calculate the column index of P and N
        int Col = blockIdx.x*blockDim.x+threadIdx.x;

        __shared__ float sum_M[TILE_WIDTH][TILE_WIDTH];
        __shared__ float sum_N[TILE_WIDTH][TILE_WIDTH];

        sum_M[threadIdx.y][threadIdx.x]=0.0;
        sum_N[threadIdx.y][threadIdx.x]=0.0;

        float Pval = 0;
        for(int k=0; k<((Width - 1)/TILE_WIDTH + 1); k++)
        {
                //printf("Col:%d, Row:%d, k:%d, th:(%d,%d), ");
                if(k*TILE_WIDTH + threadIdx.x < Width && Row < Width)
                        sum_M[threadIdx.y][threadIdx.x] = M[Row*Width + k*TILE_WIDTH + threadIdx.x];
                else sum_M[threadIdx.y][threadIdx.x] = 0.0;

                if(k*TILE_WIDTH + threadIdx.y < Width && Col < Width)
                        sum_N[threadIdx.y][threadIdx.x] = N[(k*TILE_WIDTH + threadIdx.y)*Width + Col];
                else sum_N[threadIdx.y][threadIdx.x] = 0.0;

                __syncthreads();

                for(int n=0; n<TILE_WIDTH;++n)
                        Pval += sum_M[threadIdx.y][n] * sum_N[n][threadIdx.x];

                __syncthreads();

        }
    if(Row < Width && Col < Width)
        {
                P[Row * Width + Col]  = Pval;
                //printf("(%d,%d)=%f\n",Row,Col,P[Row*Width+Col]);
        }


}

void multiply(float* M, float* N, float* P, int size) {
    float X[size][size], Y[size][size], Z[size][size];
    int i, j;

    printf("Rewriting matrices\n");
    for (i=0;i<size;i++) {
        for (j=0;j<size;j++) {
            X[i][j]=M[size*i+j];
            Y[i][j]=N[size*i+j];
            Z[i][j]=0;
        }
    }

    // Multiplying first and second matrices and storing in Z.
    for (i = 0; i < size; ++i) {
        for (j = 0; j < size; ++j) {
            for (int k = 0; k < size; ++k) {
                Z[i][j] += X[i][k] * Y[k][j];
            }
    }
    }
    printf("Result matrix:\n");
    for (i=0;i<size;i++) {
        for (j=0;j<size;j++) {
            P[size*i+j]=Z[i][j];
        }
    printf("\n");
    }


}

void matrixMultiplication(float *M, float *N, float *P, int Width){

    // declare the number of blocks per grid and the number of threads per block
    int th = TILE_WIDTH;
    int bl = (Width/TILE_WIDTH) + 1;
    dim3 threadsPerBlock(th,th,1);
    dim3 blocksPerGrid(bl,bl,1);
    printf("Kernel started: (%d,%d,1) grid, (%d,%d,1) blocks.\n", bl,bl, th,th);
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, P, Width);
}

void PrintMatrix(float* M, int Width)
{
        for(int i = 0; i < Width; i++)
        {
                for(int j = 0; j < Width; j++)
                        printf("%f  ",M[i*Width+j]);
                printf("\n");
        }
    printf("\n");
}

void matrixSingleMultiplication(float *M, float *N, float *P) {
        printf("Single threaded multiplication happening\n");
        PrintMatrix(M, MATRIX_SIZE);

}


int main(void)
{
        printf("Starting the program:\n");
        hipError_t err = hipSuccess;

        int matrix_size = MATRIX_SIZE;
        int num_of_elements = matrix_size * matrix_size;
        size_t size = num_of_elements * sizeof(float);
        printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);

    //==========================Shared Memory============================================

        //allocate matrixes on the device:
        printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
        float *M;
        err = hipMallocManaged((void**)&M,  size);
        if(err != hipSuccess)
        {
                fprintf(stderr, "Failed to allocate M matrix!\n");
                exit(EXIT_FAILURE);
        } else printf("Allocation successful.\n");

    printf("Second matrix.\n");
        float *N;
        err = hipMallocManaged((void**)&N,  size);
        if(err != hipSuccess)
        {
                fprintf(stderr, "Failed to allocate N matrix!\n");
                exit(EXIT_FAILURE);
        } else printf("Allocation successful.\n");

        printf("Third matrix.\n");
        float *P;
        err = hipMallocManaged((void**)&P,  size);
        if(err != hipSuccess)
        {
                fprintf(stderr, "Failed to allocate P matrix!\n");
                exit(EXIT_FAILURE);
        } else printf("Allocation successful.\n");

        //initialisation:
        for(int i=0; i<num_of_elements; i++)
        {
          M[i] = rand()/(float)RAND_MAX;
          N[i] = rand()/(float)RAND_MAX;
        }
    printf("Initialisation finished.\n");

    //calculations:
    clock_t start=clock();
    matrixMultiplication(M, N, P, matrix_size);
    clock_t end=clock();
    double time_elapsed=((double) (end - start)) / CLOCKS_PER_SEC;
    err = hipGetLastError();

    if(err != hipSuccess)
    {
                fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
        } else printf("Kernel operations successful. Time elapsed: %lf s.\n", time_elapsed);



    //==========================TEST===============================================
        PrintMatrix(M, matrix_size);
        PrintMatrix(N, matrix_size);
        PrintMatrix(P, matrix_size);

        for(int i = 0; i < matrix_size; i++)
        {
                for(int j = 0; j < matrix_size; j++)
                {
                        float tmp = 0;
                        for(int k = 0; k < matrix_size; k++)
                                tmp += M[i*matrix_size + k] * N[k*matrix_size + j];
                        //debug line:
                        //printf("%f ",tmp);
                        if(fabs(tmp - P[i*matrix_size + j]) > 1e-3)
                        {
                                fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
                                        i,j,tmp,P[i*matrix_size + j]);
                                exit(EXIT_FAILURE);
                        }
                }
        }

        printf("Test PASSED\n");

    //============================ Single-threaded approach ==========================

        multiply(M, N, P, MATRIX_SIZE);
        PrintMatrix(P, matrix_size);
        printf("Now freeing memory.\n");

    // Free device global memory
    err = hipFree(M);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(N);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix N (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(P);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix P (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;

}
