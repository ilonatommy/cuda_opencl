#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */
#include <stdio.h>
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if ((Row < Width) && (Col < Width)) {
		float Pvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < Width; ++k) {
			Pvalue += M[Row*Width+k]*N[k*Width+Col];
		}
		P[Row*Width+Col] = Pvalue;
	}
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = 2;
    int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================HOST===============================================

	//allocate matrixes (two input ones, one output one):
    //matrix can be represented as a flat vector in the memory - it is so in GPU, 
    //so for simplification of indexation I also use this representation on the host
	printf("Started variables allocation for the host.\n");
	float *M_h = (float *)malloc(size);
	float *N_h = (float *)malloc(size);
	float *P_h = (float *)malloc(size);

	if(M_h == NULL || N_h == NULL || P_h == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	}else printf("Allocation on host successful.\n");

	//initialize matrices:
	printf("Started initialization.\n");
	for(int i = 0; i < num_of_elements; i++)
	{
        M_h[i] = rand()/(float)RAND_MAX;
        N_h[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialization fnished.\n");

    //==========================DEVICE==============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float **M_d;
	err = hipMalloc((void**)&M_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float **N_h;
	err = hipMalloc((void**)&N_h,  size);
	if(err != hipSuccess)
	{
		printf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float **P_h;
	err = hipMalloc((void**)&P_h,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

    
    //copy marices into the device:
    printf("Started variables copying into the device.\n");
    printf("First matrix.\n");
	err = hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy first matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    printf("Second matrix.\n");
	float **N_h;
	err = hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy second matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
	
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_of_elements + threadsPerBlock - 1) / threadsPerBlock;
    printf("Kernel started: %d blocks, %d threads.\n", blocksPerGrid, threadsPerBlock);
	MatrixMulKernel <<<blocksPerGrid, threadsPerBlock>>>(M_d, N_d, P_d, matrix_size);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    printf("Started variables copying from the device.\n");
    err = hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy result matrix. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    //==========================TEST===============================================
    for(int i = 0; i < num_of_elements; i++)
	{
        if(fabs(M_h[i] * N_h[i] - P_h[i] > 1e-3)
        {
            fprintf(stderr, "Verification tests failed!\n");
            exit(EXIT_FAILURE);
        } 
	} 
	else printf("Test PASSED\n");

}
