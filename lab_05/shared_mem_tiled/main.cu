
/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixMultiplicationKernel(float* M, float* N, float* P, int Width) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	const int tile_width = 2;
	__shared__ float sum_M[tile_width][tile_width];
	__shared__ float sum_N[tile_width][tile_width];

	float Pval = 0;

	for(int k=0; k<(tile_width + Width + 1)/tile_width; k++)
	{
		if(k*tile_width + threadIdx.x < Width && Row < Width)
			sum_M[threadIdx.y][threadIdx.x] = M[Row*Width + k*tile_width + threadIdx.x];
		else sum_M[threadIdx.y][threadIdx.x] = 0.0;
		
		if(k*tile_width + threadIdx.y < Width && Col < Width)
			sum_N[threadIdx.y][threadIdx.x] = N[(k*tile_width + threadIdx.y)*Width + Col];
		else sum_M[threadIdx.y][threadIdx.x] = 0.0;

		__syncthreads();

		for(int n=0; n<tile_width;n++)
			Pval += sum_M[threadIdx.y][n] * sum_N[n][threadIdx.x];

		__syncthreads();

	}
	if(Row < Width && Col < Width)
		P[((blockIdx.y * blockDim.y + threadIdx.y) * Width) +
		(blockIdx.x * blockDim.x) + threadIdx.x] = Pval;

}

void matrixMultiplication(float *M, float *N, float *P, int Width){

    // declare the number of blocks per grid and the number of threads per block
    int threadsPerBlock = Width*Width;
    int blocksPerGrid = Width*Width;
    printf("Kernel started: %d blocks, %d threads.\n", blocksPerGrid, threadsPerBlock);
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, P, Width);
}

void PrintMatrix(float* M, int Width)
{
	for(int i = 0; i < Width; i++)
	{
		for(int j = 0; j < Width; j++)
			printf("%f  ",M[i*Width+j]);
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = 4;
    	int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================Shared Memory============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float *M;
	err = hipMallocManaged((void**)&M,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate M matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float *N;
	err = hipMallocManaged((void**)&N,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate N matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float *P;
	err = hipMallocManaged((void**)&P,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate P matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

	//initialisation:
	for(int i=0; i<num_of_elements; i++)
	{
	  M[i] = rand()/(float)RAND_MAX;
	  N[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialisation finished.\n");
    
    //calculations:
    matrixMultiplication(M, N, P, matrix_size);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    
    //==========================TEST===============================================
	PrintMatrix(M, matrix_size);
	PrintMatrix(N, matrix_size);
	PrintMatrix(P, matrix_size);	

	for(int i = 0; i < matrix_size; i++)
	{
		for(int j = 0; j < matrix_size; j++)
		{
			float tmp = 0;
			for(int k = 0; k < matrix_size; k++)
				tmp += M[i*matrix_size + k] * N[k*matrix_size + j];
			printf("%f ",tmp);
			if(fabs(tmp - P[i*matrix_size + j] > 1e-3))
			{
				fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
					i,j,tmp,P[i*matrix_size + j]);
				exit(EXIT_FAILURE);
			}
		}
    	}

	printf("Test PASSED\n");
    
    // Free device global memory
    err = hipFree(M);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(N);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix N (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(P);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix P (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Done\n");
    return 0;    
	
}

