/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void MatrixMulKernel(float *A, float *B, float *C, int N) {
	int ROW = blockIdx.y*blockDim.y+threadIdx.y;
	int COL = blockIdx.x*blockDim.x+threadIdx.x;
	float tmpCellSum=0.0f;
	if ((ROW < N) && (COL<N)) {
		for (int i = 0; i < N; i++) {
	            tmpCellSum += A[ROW * N + i] * B[i * N + COL];
		}
	}
	C[ROW * N + COL] = tmpCellSum;
}

void PrintMatrix(float* M, int Width)
{
	for(int i = 0; i < Width; i++)
	{
		for(int j = 0; j < Width; j++)
			printf("%f  ",M[i*Width+j]);
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = 3;
    	int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================HOST===============================================

	//allocate matrixes (two input ones, one output one):
    //matrix can be represented as a flat vector in the memory - it is so in GPU, 
    //so for simplification of indexation I also use this representation on the host
	printf("Started variables allocation for the host.\n");
	float *M_h = (float *)malloc(size);
	float *N_h = (float *)malloc(size);
	float *P_h = (float *)malloc(size);

	if(M_h == NULL || N_h == NULL || P_h == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	}else printf("Allocation on host successful.\n");

	//initialize matrices:
	printf("Started initialization.\n");
	for(int i = 0; i < num_of_elements; i++)
	{
        M_h[i] = rand()/(float)RAND_MAX;
        N_h[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialization fnished.\n");

    //==========================DEVICE==============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float *M_d;
	err = hipMalloc((void**)&M_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float *N_d;
	err = hipMalloc((void**)&N_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float *P_d;
	err = hipMalloc((void**)&P_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

    
    //copy marices into the device:
    printf("Started variables copying into the device.\n");
    printf("First matrix.\n");
	err = hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy first matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    printf("Second matrix.\n");
	err = hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy second matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
	
    int threadsPerBlock = 32;
    int blocksPerGrid = 32 * num_of_elements;
    printf("Kernel started: %d blocks, %d threads.\n", blocksPerGrid, threadsPerBlock);

    MatrixMulKernel <<<blocksPerGrid, threadsPerBlock>>>(M_d, N_d, P_d, matrix_size);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    printf("Started variables copying from the device.\n");
    err = hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy result matrix. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    //==========================TEST===============================================
	PrintMatrix(M_h, matrix_size);
	PrintMatrix(N_h, matrix_size);
	PrintMatrix(P_h, matrix_size);	

	for(int i = 0; i < matrix_size; i++)
	{
		for(int j = 0; j < matrix_size; j++)
		{
			float tmp = 0;
			for(int k = 0; k < matrix_size; k++)
				tmp += M_h[i*matrix_size + k] * N_h[k*matrix_size + j];
			printf("%f ",tmp);
			if(fabs(tmp - P_h[i*matrix_size + j] > 1e-3))
			{
				fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
					i,j,P_h[i*matrix_size + j],tmp);
				exit(EXIT_FAILURE);
			}
		}
    	}

	printf("Test PASSED\n");
	
}
