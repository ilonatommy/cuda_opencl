#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */
#include <stdio.h>
__global__ void matrixMultiplicationKernel(float* M, float* N, float* P, int Width) {
	// Calculate the row index of the P element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of P and N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    __shared__ float sum_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float sum_N[TILE_WIDTH][TILE_WIDTH];
	
	s_M[threadIdx.y][threadIdx.x]=M[threadIdx.y][threadIdx.x];
	s_N[threadIdx.y][threadIdx.x]=N[threadIdx.y][threadIdx.x];
    
    _syncthreads();
    
	//debug line:
	//printf("Row:%d, Col:%d. BlockIdx(%d,%d), blockDim(%d,%d) threadIdx(%d,%d)\n\n",Row,Col,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
	if ((Row < Width) && (Col < Width)) {
		float Pvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < Width; ++k) {
			Pvalue += s_M[Row*Width+k]*s_N[k*Width+Col];
		}
		P[Row*Width+Col] = Pvalue;
	}
	else P[Row*Width+Col] =  99.9;
}

void matrixMultiplication(float *M, float *N, float *P, int Width){

    // declare the number of blocks per grid and the number of threads per block
    int th = Width;
    int bl = 1;
    dim3 threadsPerBlock(th,th);
    dim3 blocksPerGrid(bl,bl);
    printf("Kernel started: %d blocks, %d threads.\n", bl, th);
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, P, Width);
}

void PrintMatrix(float* M, int Width)
{
	for(int i = 0; i < Width; i++)
	{
		for(int j = 0; j < Width; j++)
			printf("%f  ",M[i*Width+j]);
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = 10;
    	int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================HOST===============================================

	//allocate matrixes (two input ones, one output one):
    //matrix can be represented as a flat vector in the memory - it is so in GPU, 
    //so for simplification of indexation I also use this representation on the host
	printf("Started variables allocation for the host.\n");
	float *M_h = (float *)malloc(size);
	float *N_h = (float *)malloc(size);
	float *P_h = (float *)malloc(size);

	if(M_h == NULL || N_h == NULL || P_h == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	}else printf("Allocation on host successful.\n");

	//initialize matrices:
	printf("Started initialization.\n");
	for(int i = 0; i < num_of_elements; i++)
	{
        M_h[i] = rand()/(float)RAND_MAX;
        N_h[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialization fnished.\n");

    //==========================DEVICE==============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float *M_d;
	err = hipMalloc((void**)&M_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float *N_d;
	err = hipMalloc((void**)&N_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float *P_d;
	err = hipMalloc((void**)&P_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

    
    //copy marices into the device:
    printf("Started variables copying into the device.\n");
    printf("First matrix.\n");
	err = hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy first matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    printf("Second matrix.\n");
	err = hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy second matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
	
    //calculations:
    matrixMultiplication(M_d, N_d, P_d, matrix_size);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    printf("Started variables copying from the device.\n");
    err = hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy result matrix. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    //==========================TEST===============================================
	PrintMatrix(M_h, matrix_size);
	PrintMatrix(N_h, matrix_size);
	PrintMatrix(P_h, matrix_size);	

	for(int i = 0; i < matrix_size; i++)
	{
		for(int j = 0; j < matrix_size; j++)
		{
			float tmp = 0;
			for(int k = 0; k < matrix_size; k++)
				tmp += M_h[i*matrix_size + k] * N_h[k*matrix_size + j];
			//debug line:
			//printf("%f ",tmp);
			if(fabs(tmp - P_h[i*matrix_size + j]) > 1e-3)
			{
				fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
					i,j,tmp,P_h[i*matrix_size + j]);
				exit(EXIT_FAILURE);
			}
		}
    	}

	printf("Test PASSED\n");
    
    // Free device global memory
    err = hipFree(M_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(N_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix N (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(P_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix P (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // Free host memory
    free(M_h);
    free(N_h);
    free(P_h);
    
    printf("Done\n");
    return 0;    
	
}

