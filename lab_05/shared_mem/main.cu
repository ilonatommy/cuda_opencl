/*
 * main.cu
 *
 *  Created on: Nov 14, 2019
 *      Author: cuda-s01
 */

const int WIDTH = 10;


#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixMultiplicationKernel(float* M, float* N, float* P) {
    // Calculate the row index of the P element and M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column index of P and N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    
    __shared__ float s_M[WIDTH * WIDTH];
    __shared__ float s_N[WIDTH * WIDTH];
 
	//debug line:
	//printf("Row:%d, Col:%d. BlockIdx(%d,%d), blockDim(%d,%d) threadIdx(%d,%d)\n\n",Row,Col,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y,threadIdx.x,threadIdx.y);
     if ((Row < WIDTH) && (Col < WIDTH)) {
         float Pvalue = 0;
         
         s_M[Row*WIDTH + Col] = M[Row*WIDTH + Col];
         s_N[Row*WIDTH + Col] = N[Row*WIDTH + Col];
         __syncthreads();

	 // each thread computes one element of the block sub-matrix
	 for (int k = 0; k < WIDTH; ++k) {
	    Pvalue += s_M[Row*WIDTH+k]*s_N[k*WIDTH+Col];
	}
	P[Row*WIDTH+Col] = Pvalue;
      }
      else P[Row*WIDTH+Col] =  99.9;
}

void matrixMultiplication(float *M, float *N, float *P){

    // declare the number of blocks per grid and the number of threads per block
    int th = WIDTH;
    int bl = 1;
    dim3 threadsPerBlock(th,th);
    dim3 blocksPerGrid(bl,bl);
    printf("Kernel started: %d blocks, %d threads.\n", bl, th);
    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(M, N, P);
}

void PrintMatrix(float* M)
{
	for(int i = 0; i < WIDTH; i++)
	{
		for(int j = 0; j < WIDTH; j++)
			printf("%f  ",M[i*WIDTH+j]);
		printf("\n");
	}
	printf("\n");
}

int main(void)
{
	printf("Starting the program:\n");
	hipError_t err = hipSuccess;

	int matrix_size = WIDTH;
    	int num_of_elements = matrix_size * matrix_size;
	size_t size = num_of_elements * sizeof(float);
	printf("matrix [%d x %d] multiplication.\n", matrix_size, matrix_size);
    
    //==========================HOST===============================================

	//allocate matrixes (two input ones, one output one):
    //matrix can be represented as a flat vector in the memory - it is so in GPU, 
    //so for simplification of indexation I also use this representation on the host
	printf("Started variables allocation for the host.\n");
	float *M_h = (float *)malloc(size);
	float *N_h = (float *)malloc(size);
	float *P_h = (float *)malloc(size);

	if(M_h == NULL || N_h == NULL || P_h == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	}else printf("Allocation on host successful.\n");

	//initialize matrices:
	printf("Started initialization.\n");
	for(int i = 0; i < num_of_elements; i++)
	{
        M_h[i] = rand()/(float)RAND_MAX;
        N_h[i] = rand()/(float)RAND_MAX;
	}
	printf("Initialization fnished.\n");

    //==========================DEVICE==============================================
    
	//allocate matrixes on the device:
	printf("Started variables allocation for the device.\n");
    printf("First matrix.\n");
	float *M_d;
	err = hipMalloc((void**)&M_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
    
    printf("Second matrix.\n");
	float *N_d;
	err = hipMalloc((void**)&N_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");
	
	printf("Third matrix.\n");
	float *P_d;
	err = hipMalloc((void**)&P_d,  size);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate host matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Allocation successful.\n");

    
    //copy marices into the device:
    printf("Started variables copying into the device.\n");
    printf("First matrix.\n");
	err = hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy first matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    printf("Second matrix.\n");
	err = hipMemcpy(N_d, N_h, size, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy second matrix!\n");
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
	
    //calculations:
    matrixMultiplication(M_d, N_d, P_d);
    err = hipGetLastError();
    
    if(err != hipSuccess)
    {
		fprintf(stderr, "Failed to launch kernel. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Kerel operations successful.\n");
    
    printf("Started variables copying from the device.\n");
    err = hipMemcpy(P_h, P_d, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy result matrix. Error: %s.\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	} else printf("Copying successful.\n");
    
    //==========================TEST===============================================
	PrintMatrix(M_h);
	PrintMatrix(N_h);
	PrintMatrix(P_h);	

	for(int i = 0; i < matrix_size; i++)
	{
		for(int j = 0; j < matrix_size; j++)
		{
			float tmp = 0;
			for(int k = 0; k < matrix_size; k++)
				tmp += M_h[i*matrix_size + k] * N_h[k*matrix_size + j];
			//debug line:
			//printf("%f ",tmp);
			if(fabs(tmp - P_h[i*matrix_size + j]) > 1e-3)
			{
				fprintf(stderr, "Verification test failed.!\nElement at index (%d, %d) should be %f, but is %f. \n",
					i,j,tmp,P_h[i*matrix_size + j]);
				exit(EXIT_FAILURE);
			}
		}
    	}

	printf("Test PASSED\n");
    
    // Free device global memory
    err = hipFree(M_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix M (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(N_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix N (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(P_d);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix P (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // Free host memory
    free(M_h);
    free(N_h);
    free(P_h);
    
    printf("Done\n");
    return 0;    
	
}

