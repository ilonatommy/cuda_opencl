#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//#include <helper_cuda.h>
#include <time.h>
#define BLOCK_SIZE 1024
#define NUMBER_OF_ELEMENTS 1024

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void total(float *input, float *output, int len) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2* blockIdx.x * BLOCK_SIZE;
    if (start + t < len)
        partialSum[t] = input[start + t];
    else
        partialSum[t] = 0;
    if (start + BLOCK_SIZE + t < len)
        partialSum[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
    else
        partialSum[BLOCK_SIZE + t] = 0;
    //@@ Traverse the reduction tree
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
        __syncthreads();
        if (t < stride)
            partialSum[t] += partialSum[t + stride];
    }
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
    if (t == 0)
        output[blockIdx.x] = partialSum[0];
}


int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int numElements = NUMBER_OF_ELEMENTS;

    int memorySizeIn=numElements*sizeof(float);
    int memorySizeOut=sizeof(float);
    printf("Calculating the sum of %d elements.\n", numElements);
    printf("Allocating host vectors...\n");
    // Allocate the host output vector
    float *h_input = (float *)malloc(memorySizeIn);
    // Allocate the host output vector
    float *h_output = (float *)malloc(memorySizeIn);

    // Verify that allocations succeeded
    if (h_input == NULL || h_output == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    else
        printf("Success.\n");

    // Initialize the host input vector with random values
    for (int i = 0; i < numElements; ++i)
    {
        h_input[i]=(float)rand()/(float)RAND_MAX;
        // printf("%f ", h_input[i]);
    }

    printf("Allocating device vectors... \n");
    float *d_input = NULL;
    err = hipMallocManaged((void **)&d_input, memorySizeIn);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input vector!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("Device input vector allocated.\n");

    float *d_output = NULL;
    err = hipMallocManaged((void **)&d_output, memorySizeIn);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device output vector!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("Device output vector allocated.\n");

    printf("Copying input vector from the host memory to the CUDA device...\n");
    err = hipMemcpy(d_input, h_input, memorySizeIn, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input vector from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("Success.\n");

    // Launch the Kernel
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launching with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    float sumGPU = 0;
    int k = 0;
    int iter=0;

    clock_t start=clock();

    int inputElementsLeft = numElements;
    while(inputElementsLeft > 0)
    {
        int currNumElements = inputElementsLeft;
        if(inputElementsLeft > BLOCK_SIZE*2)
            currNumElements = BLOCK_SIZE*2;

        total<<<blocksPerGrid, threadsPerBlock>>>(d_input + k, d_output, currNumElements);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch the kernel in %d iteration (error code %s)!\n", iter, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        err = hipMemcpy(h_output, d_output, memorySizeOut, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy output vector from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        float sum = h_output[0];
        sumGPU+=sum;

        k+=currNumElements;
        iter+=1;
        
        inputElementsLeft-=currNumElements;
        
    }

    hipDeviceSynchronize();
    clock_t end=clock();

    double time_elapsed_gpu=((double) (end - start))*1000 / CLOCKS_PER_SEC;
    // Calculating the sum using CPU
    
    float sumCPU=0;
    for (int i=0; i<numElements; i++) {
        sumCPU+=h_input[i];
    }
    
    printf("GPU sum is %f and CPU sum is %f.\n", sumGPU, sumCPU);
    printf("Time elapsed for GPU computations: %lf ms.\n", time_elapsed_gpu);
    // Free device memory
    err = hipFree(d_input);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free input vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free output vector (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}

